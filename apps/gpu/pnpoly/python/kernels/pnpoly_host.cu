#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "pnpoly.cu"

#define max_streams 1

/*
 * This function contains the host code for benchmarking the cn_pnpoly CUDA kernel
 * Including the time spent on data transfers between host and device memory
 *
 * This host code uses device mapped host memory to overlap communication
 * between host and device with kernel execution on the GPU. Because each input
 * is read only once and each output is written only once, this implementation
 * almost fully overlaps all communication and the kernel execution time dominates
 * the total execution time.
 *
 * The code has the option to precompute all polygon line slopes on the CPU and
 * reuse those results on the GPU, instead of recomputing them on the GPU all
 * the time. The time spent on precomputing these values on the CPU is also 
 * taken into account by the time measurement in the code below. 
 *
 * This code was written for use with the Kernel Tuner. See: 
 *      https://github.com/benvanwerkhoven/kernel_tuner
 *
 * Author: Ben van Werkhoven <b.vanwerkhoven@esciencecenter.nl>
 */
float cn_pnpoly_host(int* bitmap, float2* points, float2* vertices, int n) {

    hipError_t err;
    float2 *h_vertices;
    float *h_slopes;
    float2 *h_points;
    int *h_bitmap;

    //Allocate pinned and aligned host memory and copy input data
    err = hipHostAlloc((void **)&h_vertices, VERTICES*sizeof(float2), hipHostMallocMapped);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipHostAlloc: %s\n", hipGetErrorString(err));
    }
    err = hipHostAlloc((void **)&h_slopes, VERTICES*sizeof(float), hipHostMallocMapped);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipHostAlloc: %s\n", hipGetErrorString(err));
    }
    err = hipHostAlloc((void **)&h_points, block_size_x*tile_size*grid_size_x*sizeof(float2), hipHostMallocMapped);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipHostAlloc: %s\n", hipGetErrorString(err));
    }
    err = hipHostAlloc((void **)&h_bitmap, block_size_x*tile_size*grid_size_x*sizeof(int), hipHostMallocMapped);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipHostAlloc: %s\n", hipGetErrorString(err));
    }
    memcpy(h_vertices, vertices, VERTICES*sizeof(float2));
    memcpy(h_points, points, n*sizeof(float2));

    //create CUDA streams and events
    hipStream_t stream[max_streams];
    for (int i=0; i<max_streams; i++) {
        err = hipStreamCreate(&stream[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "Error in hipStreamCreate: %s\n", hipGetErrorString(err));
        }
    }
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipEventCreate: %s\n", hipGetErrorString(err));
    }

    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipEventCreate: %s\n", hipGetErrorString(err));
    }

    //kernel parameters
    dim3 threads(block_size_x, block_size_y, block_size_z);
    dim3 grid(grid_size_x, grid_size_y);

    //start measuring time
    hipDeviceSynchronize();
    hipEventRecord(start, stream[0]);

    //transfer vertices to d_vertices
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_vertices), h_vertices, VERTICES*sizeof(float2), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }

    #if use_precomputed_slopes == 1
    //precompute the slopes and transfer to symbol d_slopes
    h_slopes[0] = (h_vertices[VERTICES-1].x - h_vertices[0].x) / (h_vertices[VERTICES-1].y - h_vertices[0].y);
    for (int i=1; i<VERTICES; i++) {
        h_slopes[i] = (h_vertices[i-1].x - h_vertices[i].x) / (h_vertices[i-1].y - h_vertices[i].y);
    }
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_slopes), h_slopes, VERTICES*sizeof(float), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }
    #endif

    //call the kernel
    cn_pnpoly<<<grid, threads, 0, stream[0]>>>(h_bitmap, h_points, n);  //using mapped memory

    //stop time measurement
    hipEventRecord(stop, stream[0]);
    hipDeviceSynchronize();
    float time = 0.0;
    hipEventElapsedTime(&time, start, stop);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Cuda error after kernel: %s.\n", hipGetErrorString(err));
    }

    //copy data back to output parameter for correctness checking
    memcpy(bitmap, h_bitmap, n*sizeof(int));

    //cleanup
    hipHostFree(h_points);
    hipHostFree(h_vertices);
    hipHostFree(h_slopes);
    hipHostFree(h_bitmap);
    for (int i=0; i<max_streams; i++) {
        err = hipStreamDestroy(stream[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time; //ms
}
