#include "hip/hip_runtime.h"
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include "g_pnpoly.h"
}

#define VERTICES 2000

#ifndef between_method
#define between_method 1
#endif

#ifndef use_method
#define use_method 1
#endif

#ifndef block_size_x
#define block_size_x 256
#endif
#ifndef block_size_y
#define block_size_y 1
#endif
#ifndef block_size_z
#define block_size_z 1
#endif

#ifndef tile_size
#define tile_size 1
#endif

#define max_streams 1
__constant__ float d_verticesX[VERTICES];
__constant__ float d_verticesY[VERTICES];
__constant__ float d_slopes[VERTICES];

__device__ __forceinline__ int is_between(float a, float b, float c) {
    #if between_method == 0
        return (b > a) != (c > a);
    #elif between_method == 1
        return ((b <= a) && (c > a)) || ((b > a) && (c <= a));
    #elif between_method == 2
        return ((a - b) == 0.0f) || ((a - b) * (a - c) < 0.0f);
    #elif between_method == 3
        //Interestingly enough method 3 exactly the same as method 2, only in a different order.
        //the performance difference between method 2 and 3 can be huge depending on all the other optimization parameters.
        return ((a - b) * (a - c) < 0.0f) || (a - b == 0.0f);
    #endif
}

__global__ void cn_pnpolyBEN(char* bitmap, float *px, float *py, int npoints, int nverts) {
    int i = blockIdx.x * block_size_x * tile_size + threadIdx.x;
    if (i < npoints) {

        char c[tile_size];
        float2 lpoints[tile_size];
        #pragma unroll
        for (int ti=0; ti<tile_size; ti++) {
            c[ti] = 0;
            lpoints[ti] = make_float2(px[i+block_size_x*ti], py[i+block_size_x*ti]);
        }

        int k = nverts-1;

        for (int j=0; j<nverts; k = j++) {    // edge from vj to vk
            float2 vj = make_float2(d_verticesX[j], d_verticesY[j]); 
            float2 vk = make_float2(d_verticesX[k], d_verticesY[k]); 

            #if use_precomputed_slopes == 0
            float slope = (vk.x-vj.x) / (vk.y-vj.y);
            #elif use_precomputed_slopes == 1
            float slope = d_slopes[j];
            #endif

            #pragma unroll
            for (int ti=0; ti<tile_size; ti++) {

                float2 p = lpoints[ti];

                #if use_method == 0
                if ( is_between(p.y, vj.y, vk.y) &&         //if p is between vj and vk vertically
                     (p.x < slope * (p.y-vj.y) + vj.x) ) {  //if p.x crosses the line vj-vk when moved in positive x-direction
                    c[ti] = !c[ti];
                }

                #elif use_method == 1
                //Same as method 0, but attempts to reduce divergence by avoiding the use of an if-statement.
                //Whether this is more efficient is data dependent because there will be no divergence using method 0, when none
                //of the threads within a warp evaluate is_between as true
                int b = is_between(p.y, vj.y, vk.y);
                c[ti] += b && (p.x < vj.x + slope * (p.y - vj.y));

                #endif


            }

        }

        #pragma unroll
        for (int ti=0; ti<tile_size; ti++) {
            //could do an if statement here if 1s are expected to be rare
            #if use_method == 0
            bitmap[i+block_size_x*ti] = c[ti];
            #elif use_method == 1
            bitmap[i+block_size_x*ti] = c[ti] & 1;
            #endif
        }
    }
}


// Cuda version
__global__ void pnpoly_cnGPU(char *cs, const float *px, const float *py, const float *vx, const float *vy, int npoint, int nvert)
{
    extern __shared__ int s[];
    float *tvx = (float*) s;
   	float *tvy = (float*)&s[nvert];

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k, c = 0;
    	for (j = 0, k = nvert-1; j < nvert; k = j++) {
        	tvx[j] = vx [j];
        	tvy[j] = vy [j];
    	}

    	__syncthreads();

        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((tvy[j]>py[i]) != (tvy[k]>py[i])) &&
                    (px[i] < (tvx[k]-tvx[j]) * (py[i]-tvy[j]) / (tvy[k]-tvy[j]) + tvx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }
}

__global__ void pnpoly_cnGPU1(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>py[i]) != (vy[k]>py[i])) &&
                    (px[i] < (vx[k]-vx[j]) * (py[i]-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }
}

__global__ void pnpoly_cnGPU2(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    __shared__ float tpx;
    __shared__ float tpy;
    if (i < npoint) {
        tpx = px[i];
        tpy = py[i];
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>tpy) != (vy[k]>tpy)) &&
                    (tpx < (vx[k]-vx[j]) * (tpy-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
        __syncthreads();
    }
}

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}


extern "C"

int setup_GPU() {
    int idev = 0;                                   // use default device 0
    hipDeviceProp_t deviceProp;

	//GPU_SETUP = 0;

    /*
     * if GPU found supports SM 1.2, then continue, otherwise we exit
	*/
    if (!checkCudaCapabilities(1, 2)) {
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipSetDevice(idev));

    /* Verify the selected device supports mapped memory and set the device
       flags for mapping host memory. */
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
    if (!deviceProp.canMapHostMemory) {
        fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    hipDeviceReset();
    exit(EXIT_SUCCESS);
#endif

#if CUDART_VERSION < 4000
    if (bPinGenericMemory) {
        fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
#endif

	//GPU_SETUP = 1;

	return 0;
}

void reset_GPU() {
    hipDeviceReset();
}

int pnpoly_GPU(signed char **mc, int nvert, int npoint, float *mpx, float *mpy, float *mvx, float *mvy) {
    /*GPU*/
    bool bPinGenericMemory = false;                 // Allocate generic memory with malloc() and pin it later instead of using hipHostAlloc()
    unsigned int flags;
    size_t pbytes, vbytes, cbytes;
    float *px, *py, *vx, *vy;
    float *d_px, *d_py, *d_vx, *d_vy;              // Device pointers for mapped memory
    char *c, *c_UA, *d_c;                            // Device pointers for mapped memory
    struct timeval stop, start;
    unsigned long long t;
    int i, count = 0;
    float *h_slopes;
    hipError_t err;

    //create CUDA streams and events
    hipStream_t stream[max_streams];
    for (int i=0; i<max_streams; i++) {
        err = hipStreamCreate(&stream[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "Error in hipStreamCreate: %s\n", hipGetErrorString(err));
        }
    }
    /*CUDA monitoring*/
    hipEvent_t cstart, cstop;

    if (bPinGenericMemory) {
	/*Return str with the error*/
        printf("> Using Generic System Paged Memory (malloc)\n");
    } else {
	/*Return str with the error*/
        printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
    }

    pbytes = npoint*sizeof(float);
    vbytes = nvert*sizeof(float);
    cbytes = npoint*sizeof(char);
    printf("Bytes allocated for npoints %d and nvert %d: pbytes %zu, vbytes %zu, cbytes %zu\n", npoint, nvert, pbytes, vbytes, cbytes);

    err = hipHostAlloc((void **)&h_slopes, nvert*sizeof(float), hipHostMallocMapped);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipHostAlloc: %s\n", hipGetErrorString(err));
    }

    if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
        gettimeofday(&start, NULL);
        c_UA = (char *) malloc(cbytes + MEMORY_ALIGNMENT);
        c = (char *) ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

        px = mpx;
        py = mpy;
        vx = mvx;
        vy = mvy;

        checkCudaErrors(hipHostRegister(px, pbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(py, pbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(vx, vbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(vy, vbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(c, cbytes, hipHostMallocMapped));
        gettimeofday(&stop, NULL);
        t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
        printf("PinGenericMemory took %llu ms\n", t);
#endif
    } else {
#if CUDART_VERSION >= 2020
        flags = hipHostMallocMapped;
        checkCudaErrors(hipHostAlloc((void **)&px, pbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&py, pbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&vx, vbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&vy, vbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&c, cbytes, flags));

		/*Copy point and vertices*/
    	gettimeofday(&start, NULL);
		memcpy(px, mpx, pbytes); 
		memcpy(py, mpy, pbytes); 
		memcpy(vx, mvx, vbytes); 
		memcpy(vy, mvy, vbytes); 
    	gettimeofday(&stop, NULL);
    	t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    	printf("MemCopy took %llu ms\n", t);

#endif
    }

    //transfer vertices to d_vertices
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_verticesX), vx, nvert*sizeof(float), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_verticesY), vy, nvert*sizeof(float), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }

    #if use_precomputed_slopes == 1
    //precompute the slopes and transfer to symbol d_slopes
    h_slopes[0] = (vx[nvert-1] - vx[0]) / (vy[nvert-1] - vy[0]);
    for (int i=1; i<nvert; i++) {
        h_slopes[i] = (vx[i-1] - vx[i]) / (vy[i-1] - vy[i]);
    }
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_slopes), h_slopes, nvert*sizeof(float), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }
    #endif

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */
#if CUDART_VERSION >= 2020
    gettimeofday(&start, NULL);
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_px, (void *)px, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_py, (void *)py, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_vx, (void *)vx, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_vy, (void *)vy, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Get devie pointers took %llu ms\n", t);
#endif

    /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */
    printf("> pnpoly_GPU kernel will check which points are in the Polygon using mapped CPU memory...\n");
    //dim3 block(256);
    dim3 block(BLOCK_SIZE);
    dim3 grid((unsigned int)ceil(npoint/(float)block.x));

    hipEventCreate(&cstart);
    hipEventCreate(&cstop);
    hipEventRecord(cstart, stream[0]);
    //size_t sh_size = 2*nvert*sizeof(float);
    //pnpoly_cnGPU<<<grid, block, sh_size>>>(d_c, d_px, d_py, d_vx, d_vy, npoint, nvert);

    dim3 threads(block_size_x, block_size_y, block_size_z);
	cn_pnpolyBEN<<<grid, threads, 0, stream[0]>>>(d_c, d_px, d_py, npoint, nvert);
    hipEventRecord(cstop, stream[0]);

    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("pnpoly_cnGPU() execution failed");

    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;

    hipEventSynchronize(cstop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, cstart, cstop);

    printf("PnPoly took %llu ms and %f msecs CUDA\n", t, milliseconds);

    gettimeofday(&start, NULL);
    /* Output results */
	for (i = 0; i < cbytes; i++) {
		char is = c[i];
		if (is == 1)
			count++;
	}
	printf("It has %d\n", count);
    memcpy(*mc, c, cbytes);

    /* Memory clean up */
    printf("> Releasing CPU memory...\n");

    if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
        checkCudaErrors(hipHostUnregister(px));
        checkCudaErrors(hipHostUnregister(py));
        checkCudaErrors(hipHostUnregister(vx));
        checkCudaErrors(hipHostUnregister(vy));
        checkCudaErrors(hipHostUnregister(c));
        free(c_UA);
#endif
    } else {
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostFree(px));
        checkCudaErrors(hipHostFree(py));
        checkCudaErrors(hipHostFree(vx));
        checkCudaErrors(hipHostFree(vy));
        checkCudaErrors(hipHostFree(c));
#endif
    }
    hipHostFree(h_slopes);
    for (int i=0; i<max_streams; i++) {
        err = hipStreamDestroy(stream[i]);
    }
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Output results %llu ms\n", t);

    return 0;
}

