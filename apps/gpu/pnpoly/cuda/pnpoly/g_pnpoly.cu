#include "hip/hip_runtime.h"
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include "g_pnpoly.h"
}

// Cuda version
__global__ void pnpoly_cnGPU(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    extern __shared__ int s[];
    float *tvx = (float*) s;
   	float *tvy = (float*)&s[nvert];

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k, c = 0;
    	for (j = 0, k = nvert-1; j < nvert; k = j++) {
        	tvx[j] = vx [j];
        	tvy[j] = vy [j];
    	}

    	__syncthreads();

        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((tvy[j]>py[i]) != (tvy[k]>py[i])) &&
                    (px[i] < (tvx[k]-tvx[j]) * (py[i]-tvy[j]) / (tvy[k]-tvy[j]) + tvx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }
}

__global__ void pnpoly_cnGPU1(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>py[i]) != (vy[k]>py[i])) &&
                    (px[i] < (vx[k]-vx[j]) * (py[i]-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }
}

__global__ void pnpoly_cnGPU2(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    __shared__ float tpx;
    __shared__ float tpy;
    if (i < npoint) {
        tpx = px[i];
        tpy = py[i];
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>tpy) != (vy[k]>tpy)) &&
                    (tpx < (vx[k]-vx[j]) * (tpy-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
        __syncthreads();
    }
}

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}


extern "C"

int setup_GPU() {
    int idev = 0;                                   // use default device 0
    hipDeviceProp_t deviceProp;

	//GPU_SETUP = 0;

    /*
     * if GPU found supports SM 1.2, then continue, otherwise we exit
	*/
    if (!checkCudaCapabilities(1, 2)) {
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipSetDevice(idev));

    /* Verify the selected device supports mapped memory and set the device
       flags for mapping host memory. */
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
    if (!deviceProp.canMapHostMemory) {
        fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    hipDeviceReset();
    exit(EXIT_SUCCESS);
#endif

#if CUDART_VERSION < 4000
    if (bPinGenericMemory) {
        fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
#endif

	//GPU_SETUP = 1;

	return 0;
}

void reset_GPU() {
    hipDeviceReset();
}

int pnpoly_GPU(signed char **mc, int nvert, int npoint, float *mpx, float *mpy, float *mvx, float *mvy) {
    /*GPU*/
    bool bPinGenericMemory = true;                 // Allocate generic memory with malloc() and pin it later instead of using hipHostAlloc()
    unsigned int flags;
    size_t pbytes, vbytes, cbytes;
    float *px, *py, *vx, *vy;
    //float *px_UA, *py_UA, *vx_UA, *vy_UA;          // Non-4K Aligned Pinned memory on the CPU
    float *d_px, *d_py, *d_vx, *d_vy;              // Device pointers for mapped memory
    char *c, *c_UA, *d_c;                            // Device pointers for mapped memory
    struct timeval stop, start;
    unsigned long long t;
    int i, count = 0;

    /*CUDA monitoring*/
    hipEvent_t cstart, cstop;

	/*
	if (GPU_SETUP != 1) {
        printf("> GPU_SETUP was not initialized.\n");
		setup_GPU();
	}
	*/

    if (bPinGenericMemory) {
	/*Return str with the error*/
        printf("> Using Generic System Paged Memory (malloc)\n");
    } else {
	/*Return str with the error*/
        printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
    }

    pbytes = npoint*sizeof(float);
    vbytes = nvert*sizeof(float);
    cbytes = npoint*sizeof(char);
    printf("Bytes allocated for npoints %d and nvert %d: pbytes %zu, vbytes %zu, cbytes %zu\n", npoint, nvert, pbytes, vbytes, cbytes);

    if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
        gettimeofday(&start, NULL);
        /*
        px_UA = (float *) malloc(pbytes + MEMORY_ALIGNMENT);
        py_UA = (float *) malloc(pbytes + MEMORY_ALIGNMENT);
        vx_UA = (float *) malloc(vbytes + MEMORY_ALIGNMENT);
        vy_UA = (float *) malloc(vbytes + MEMORY_ALIGNMENT);
        c_UA = (char *) malloc(cbytes + MEMORY_ALIGNMENT);

        // We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
        px = (float *) ALIGN_UP(px_UA, MEMORY_ALIGNMENT);
        py = (float *) ALIGN_UP(py_UA, MEMORY_ALIGNMENT);
        vx = (float *) ALIGN_UP(vx_UA, MEMORY_ALIGNMENT);
        vy = (float *) ALIGN_UP(vy_UA, MEMORY_ALIGNMENT);
        */
        c_UA = (char *) malloc(cbytes + MEMORY_ALIGNMENT);
        c = (char *) ALIGN_UP(c_UA, MEMORY_ALIGNMENT);
		/*
        px = (float *) ALIGN_UP(mpx, MEMORY_ALIGNMENT);
        py = (float *) ALIGN_UP(mpy, MEMORY_ALIGNMENT);
        vx = (float *) ALIGN_UP(mvx, MEMORY_ALIGNMENT);
        vy = (float *) ALIGN_UP(mvy, MEMORY_ALIGNMENT);
        */

        px = mpx;
        py = mpy;
        vx = mvx;
        vy = mvy;

        checkCudaErrors(hipHostRegister(px, pbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(py, pbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(vx, vbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(vy, vbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(c, cbytes, hipHostMallocMapped));
        gettimeofday(&stop, NULL);
        t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
        printf("PinGenericMemory took %llu ms\n", t);
#endif
    } else {
#if CUDART_VERSION >= 2020
        flags = hipHostMallocMapped;
        checkCudaErrors(hipHostAlloc((void **)&px, pbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&py, pbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&vx, vbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&vy, vbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&c, cbytes, flags));
#endif
    }

	/*Copy point and vertices*/
    gettimeofday(&start, NULL);
	//memcpy(px, mpx, pbytes); 
	//memcpy(py, mpy, pbytes); 
	//memcpy(vx, mvx, vbytes); 
	//memcpy(vy, mvy, vbytes); 
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("MemCopy took %llu ms\n", t);

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */
#if CUDART_VERSION >= 2020
    gettimeofday(&start, NULL);
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_px, (void *)px, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_py, (void *)py, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_vx, (void *)vx, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_vy, (void *)vy, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Get devie pointers took %llu ms\n", t);
#endif

    /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */
    printf("> pnpoly_GPU kernel will check which points are in the Polygon using mapped CPU memory...\n");
    //dim3 block(256);
    dim3 block(BLOCK_SIZE);
    dim3 grid((unsigned int)ceil(npoint/(float)block.x));
    //vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
    //gettimeofday(&start, NULL);
    
    //pnpoly_cnGPU<<<grid, block>>>(d_px, d_py, d_vx, d_vy, d_c, npoint, nvert);
    //size_t sh_size = 2*607*sizeof(float);
    //pnpoly_cnGPU<<<grid, block,sh_size>>>(d_px, d_py, d_vx, d_vy, d_c, npoint, nvert);

    hipEventCreate(&cstart);
    hipEventCreate(&cstop);
    hipEventRecord(cstart);
    size_t sh_size = 2*nvert*sizeof(float);
    pnpoly_cnGPU<<<grid, block, sh_size>>>(d_px, d_py, d_vx, d_vy, d_c, npoint, nvert);
    hipEventRecord(cstop);

    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("pnpoly_cnGPU() execution failed");

    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;

    hipEventSynchronize(cstop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, cstart, cstop);

    printf("PnPoly took %llu ms and %f msecs CUDA\n", t, milliseconds);

    gettimeofday(&start, NULL);
    /* Output results */
	for (i = 0; i < cbytes; i++) {
		char is = c[i];
		if (is == 1)
			count++;
	}
	printf("It has %d\n", count);
    memcpy(*mc, c, cbytes);

    /* Memory clean up */
    printf("> Releasing CPU memory...\n");

    if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
        checkCudaErrors(hipHostUnregister(px));
        checkCudaErrors(hipHostUnregister(py));
        checkCudaErrors(hipHostUnregister(vx));
        checkCudaErrors(hipHostUnregister(vy));
        checkCudaErrors(hipHostUnregister(c));
    //    free(px_UA);
    //    free(py_UA);
    //    free(vx_UA);
    //    free(vy_UA);
        free(c_UA);
#endif
    } else {
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostFree(px));
        checkCudaErrors(hipHostFree(py));
        checkCudaErrors(hipHostFree(vx));
        checkCudaErrors(hipHostFree(vy));
        checkCudaErrors(hipHostFree(c));
#endif
    }
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Output results %llu ms\n", t);

    return 0;
}

