#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX 
#define MAX(a,b) (a > b ? a : b) 
#endif

#define THREADS_PER_BLOCK 512
#define BLOCK_SIZE 256

#define hipDeviceScheduleBlockingSync   0x04 

int pnpoly_cn(char **res, int nvert, float *vx, float *vy, int npoint, float *px, float *py)
{
    int i = 0;
    char *cs = NULL;
    cs = *res;

    for (i = 0; i < npoint; i++) {
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>py[i]) != (vy[k]>py[i])) &&
                    (px[i] < (vx[k]-vx[j]) * (py[i]-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }

    return 0;
}

float isLeft( float P0x, float P0y, float P1x, float P1y, float P2x, float P2y)
{
    return ( (P1x - P0x) * (P2y - P0y) - (P2x -  P0x) * (P1y - P0y) );
}

int pnpoly_wn(char **res, int nvert, float *vx, float *vy, int npoint, float *px, float *py)
{
    int i = 0, j =0;
    char *cs = NULL;
    cs = *res;

    for (i = 0; i < npoint; i++) {
        int wn = 0;
        for (j = 0; j < nvert-1; j++) {
            if (vy[j] <= py[i]) {
                if (vy[j+1] > py[i])
                    if (isLeft( vx[j], vy[j], vx[j+1], vy[j+1], px[i], py[i]) > 0)
                        ++wn;
            }
            else {
                if (vy[j+1]  <= py[i])
                    if (isLeft( vx[j], vy[j], vx[j+1], vy[j+1], px[i], py[i]) < 0)
                        --wn;
            }
        }
        cs[i] = wn & 1;
        //cs[i] = wn;
    }

    return 0;
}

int pnpoly_wnLeft(int **res, int nvert, float *vx, float *vy, int npoint, float *px, float *py)
{
    int i = 0, j =0;
    int *cs = NULL;
    cs = *res;

    for (i = 0; i < npoint; i++) {
        int wn = 0;
        for (j = 0; j < nvert-1; j++) {
            if (vy[j] <= py[i]) {
                if (vy[j+1] > py[i])
                    //if (isLeft( vx[j], vy[j], vx[j+1], vy[j+1], px[i], py[i]) > 0)
                    if (( (vx[j+1] - vx[j]) * (py[i] - vy[j]) - (px[i] -  vx[j]) * (vy[j+1] - vy[j]) ) > 0)
                        ++wn;
            }
            else {
                if (vy[j+1]  <= py[i])
                    //if (isLeft( vx[j], vy[j], vx[j+1], vy[j+1], px[i], py[i]) < 0)
                    if (( (vx[j+1] - vx[j]) * (py[i] - vy[j]) - (px[i] -  vx[j]) * (vy[j+1] - vy[j]) ) < 0)
                        --wn;
            }
        }
        cs[i] = wn & 1;
        //cs[i] = wn;
    }

    return 0;
}

int getPoints(char* filename, int npoint, float **px, float **py) {
    FILE *fp = NULL;
    char * line = NULL;
    size_t len = 0;
    ssize_t read = 0;
    int points = 0;
    float *ptx, *pty;

    fp = fopen(filename, "r");

    if (fp == NULL)
        exit(EXIT_FAILURE);

    ptx = *px;
    pty = *py;

    while ((read = getline(&line, &len, fp)) != -1) {
        line[read-1]='\0';
        sscanf(line, "%f %f", &ptx[points], &pty[points]);
        points++;
    }

    fclose(fp);
    if (line)
        free(line);
    if (npoint != points)
        points = 0;

    return points;
}

int outputResult(char *filename, char *cs, int npoint, float *px, float *py) {
    int i = 0;
    FILE *fp = NULL;
    fp = fopen(filename, "w");

    for (i=0; i<npoint; i++) {
        if (cs[i])
            fprintf(fp,"%lf %lf\n", px[i], py[i]);
    }
    fclose(fp);
    return 0;
}

// Modification for the structure, knowing that the first vertex is repeated in the last position of the array
int pnpoly2(int nvert, float *vertex, float testx, float testy)
{
	int i, j, c = 0;
	for (i = 1, j = i-1; i < nvert; j = i++) {
		if ( ((vertex[3*i+1]>testy) != (vertex[3*j+1]>testy)) &&
			(testx < (vertex[3*j]-vertex[3*i]) * (testy-vertex[3*i+1]) / (vertex[3*j+1]-vertex[3*i+1]) + vertex[3*i]) )
			c = !c;
	}
	return c;
}

// Cuda version
__global__ void pnpoly_cnGPU(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    __shared__ float tvx[607];
    __shared__ float tvy[607];

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            tvx[j] = vx [j];
            tvy[j] = vy [j];
            if ( ((tvy[j]>py[i]) != (tvy[k]>py[i])) &&
                    (px[i] < (tvx[k]-tvx[j]) * (py[i]-tvy[j]) / (tvy[k]-tvy[j]) + tvx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }
    __syncthreads();
}

__global__ void pnpoly_cnGPU1(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>py[i]) != (vy[k]>py[i])) &&
                    (px[i] < (vx[k]-vx[j]) * (py[i]-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
    }
}

__global__ void pnpoly_cnGPU2(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    __shared__ float tpx;
    __shared__ float tpy;
    if (i < npoint) {
        tpx = px[i];
        tpy = py[i];
        int j, k, c = 0;
        for (j = 0, k = nvert-1; j < nvert; k = j++) {
            if ( ((vy[j]>tpy) != (vy[k]>tpy)) &&
                    (tpx < (vx[k]-vx[j]) * (tpy-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
                c = !c;
        }
        cs[i] = c & 1;
        __syncthreads();
    }
}

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}



/*function [bool_out] = inpolygon_for_gpu(testx,testy,vertx,verty)


ind1=1;
nvert=length(vertx);
ind2=nvert-1;
bool_out=zeros(size(testx));
while ind1<nvert
        
            bools_to_change=find( ((verty(ind1)>testy) ~= (verty(ind2)>testy)) & ...
                (testx < (vertx(ind2)-vertx(ind1)) * ...
                    (testy-verty(ind1)) / (verty(ind2)-verty(ind1)) + vertx(ind1) ));

                bool_out(bools_to_change)=~bool_out(bools_to_change);
                    ind2=ind1;
                        ind1=ind1+1;
                        end
*/


// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

int main(int argc, char* argv[]){
    int nvert, npoint;
    struct timeval stop, start;
    unsigned long long t;

    /*GPU*/
    int idev = 0;                                   // use default device 0
    bool bPinGenericMemory = true;                 // Allocate generic memory with malloc() and pin it later instead of using hipHostAlloc()
    hipDeviceProp_t deviceProp;
    unsigned int flags;
    size_t pbytes, vbytes, cbytes;
    float *px, *py, *vx, *vy;                  // Pinned memory allocated on the CPU
    float *px_UA, *py_UA, *vx_UA, *vy_UA;          // Non-4K Aligned Pinned memory on the CPU
    float *d_px, *d_py, *d_vx, *d_vy;              // Device pointers for mapped memory
    char *c, *c_UA, *d_c;                            // Device pointers for mapped memory

    /*CUDA monitoring*/
    hipEvent_t cstart, cstop;

    if (argc != 7) {
        printf("Wrong number of arguments:\n./pnpoly <func [0 for cn | 1 for wn | 2 for wnLeft]> <points_filename> <num_points> <polygon_filename> <num_vertex> <out_filename>\n");
        return 0;
    }

    // if GPU found supports SM 1.2, then continue, otherwise we exit 
    if (!checkCudaCapabilities(1, 2)) {
        exit(EXIT_SUCCESS);
    }

    if (bPinGenericMemory) {
        printf("> Using Generic System Paged Memory (malloc)\n");
    } else {
        printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
    }

    checkCudaErrors(hipSetDevice(idev));

    /* Verify the selected device supports mapped memory and set the device
       flags for mapping host memory. */
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
    if (!deviceProp.canMapHostMemory) {
        fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    hipDeviceReset();
    exit(EXIT_SUCCESS);
#endif

#if CUDART_VERSION < 4000
    if (bPinGenericMemory) {
        fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
#endif

    /* Allocate mapped CPU memory. */
    npoint = atoi(argv[3]);
    nvert = atoi(argv[5]);
    pbytes = npoint*sizeof(float);
    vbytes = nvert*sizeof(float);
    cbytes = npoint*sizeof(char);

    if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
        gettimeofday(&start, NULL);
        px_UA = (float *) malloc(pbytes + MEMORY_ALIGNMENT);
        py_UA = (float *) malloc(pbytes + MEMORY_ALIGNMENT);
        vx_UA = (float *) malloc(vbytes + MEMORY_ALIGNMENT);
        vy_UA = (float *) malloc(vbytes + MEMORY_ALIGNMENT);
        c_UA = (char *) malloc(cbytes + MEMORY_ALIGNMENT);

        // We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
        px = (float *) ALIGN_UP(px_UA, MEMORY_ALIGNMENT);
        py = (float *) ALIGN_UP(py_UA, MEMORY_ALIGNMENT);
        vx = (float *) ALIGN_UP(vx_UA, MEMORY_ALIGNMENT);
        vy = (float *) ALIGN_UP(vy_UA, MEMORY_ALIGNMENT);
        c = (char *) ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

        checkCudaErrors(hipHostRegister(px, pbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(py, pbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(vx, vbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(vy, vbytes, hipHostMallocMapped));
        checkCudaErrors(hipHostRegister(c, cbytes, hipHostMallocMapped));
        gettimeofday(&stop, NULL);
        t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
        printf("PinGenericMemory took %llu ms\n", t);
#endif
    } else {
#if CUDART_VERSION >= 2020
        flags = hipHostMallocMapped;
        checkCudaErrors(hipHostAlloc((void **)&px, pbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&py, pbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&vx, vbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&vy, vbytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&c, cbytes, flags));
#endif
    }

    /*Points*/
    gettimeofday(&start, NULL);
    if (!getPoints(argv[2], npoint, &px, &py)) {
        //TODO clean memory
        printf("Failed to get Points!!!\n");
        return -1;
    }
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Populate Points took %llu ms\n", t);

    /*Vertex of the Polygon*/
    gettimeofday(&start, NULL);
    if (!getPoints(argv[4], nvert, &vx, &vy)) {
        //TODO clean memory
        printf("Failed to get Polygon!!!\n");
        return -1;
    }
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Populate Polygon took %llu ms\n", t);

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */
#if CUDART_VERSION >= 2020
    gettimeofday(&start, NULL);
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_px, (void *)px, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_py, (void *)py, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_vx, (void *)vx, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_vy, (void *)vy, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;
    printf("Get devie pointers took %llu ms\n", t);
#endif

    /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */
    printf("> pnpoly_cnGPU kernel will check which points are in the Polygon using mapped CPU memory...\n");
    //dim3 block(256);
    dim3 block(BLOCK_SIZE);
    dim3 grid((unsigned int)ceil(npoint/(float)block.x));
    //vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
    gettimeofday(&start, NULL);
    
    //pnpoly_cnGPU<<<grid, block>>>(d_px, d_py, d_vx, d_vy, d_c, npoint, nvert);
    //size_t sh_size = 2*607*sizeof(float);
    //pnpoly_cnGPU<<<grid, block,sh_size>>>(d_px, d_py, d_vx, d_vy, d_c, npoint, nvert);

    hipEventCreate(&cstart);
    hipEventCreate(&cstop);
    hipEventRecord(cstart);
    pnpoly_cnGPU1<<<grid, block>>>(d_px, d_py, d_vx, d_vy, d_c, npoint, nvert);
    hipEventRecord(cstop);

    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("pnpoly_cnGPU() execution failed");

    gettimeofday(&stop, NULL);
    t = 1000 * (stop.tv_sec - start.tv_sec) + (stop.tv_usec - start.tv_usec) / 1000;

    hipEventSynchronize(cstop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, cstart, cstop);

    printf("PnPoly took %llu ms and %f msecs CUDA\n", t, milliseconds);


    /* Output results */
    printf("> Checking the results from vectorAddGPU() ...\n");
    outputResult(argv[6], c, npoint, px, py);

    /* Memory clean up */
    printf("> Releasing CPU memory...\n");

    if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
        checkCudaErrors(hipHostUnregister(px));
        checkCudaErrors(hipHostUnregister(py));
        checkCudaErrors(hipHostUnregister(vx));
        checkCudaErrors(hipHostUnregister(vy));
        checkCudaErrors(hipHostUnregister(c));
        free(px_UA);
        free(py_UA);
        free(vx_UA);
        free(vy_UA);
        free(c_UA);
#endif
    } else {
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostFree(px));
        checkCudaErrors(hipHostFree(py));
        checkCudaErrors(hipHostFree(vx));
        checkCudaErrors(hipHostFree(vy));
        checkCudaErrors(hipHostFree(c));
#endif
    }
    hipDeviceReset();
}
